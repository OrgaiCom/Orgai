#include "hip/hip_runtime.h"
﻿/*
    The MIT License

    Copyright(c) 2024 OrgaiCom

    以下に定める条件に従い、本ソフトウェアおよび関連文書のファイル（以下「ソフトウェア」）
    の複製を取得するすべての人に対し、ソフトウェアを無制限に扱うことを無償で許可します。
    これには、ソフトウェアの複製を使用、複写、変更、結合、掲載、頒布、サブライセンス、
    および/または販売する権利、およびソフトウェアを提供する相手に同じことを許可する権利も
    無制限に含まれます。

    上記の著作権表示および本許諾表示を、ソフトウェアのすべての複製または重要な部分に
    記載するものとします。

    ソフトウェアは「現状のまま」で、明示であるか暗黙であるかを問わず、何らの保証もなく
    提供されます。ここでいう保証とは、商品性、特定の目的への適合性、および権利非侵害に
    ついての保証も含みますが、それに限定されるものではありません。 作者または著作権者は、
    契約行為、不法行為、またはそれ以外であろうと、ソフトウェアに起因または関連し、
    あるいはソフトウェアの使用またはその他の扱いによって生じる一切の請求、損害、
    その他の義務について何らの責任も負わないものとします。
*/

//クロック
//for iTrainDataNum Clock : 2907097002
//  入力層にパラメータを渡す Clock :    19753
//  出力値を計算する Clock :            2042214
//      入力層の各ニューロンの y の値を計算する。 Clock :           19815
//      中間層の各ニューロンに前列から y の値を伝達させつつ Clock : 1995285
//      y の値を出力層のニューロンに伝達させる Clock :              16522
//      出力層のニューロンの y の値を計算する Clock :               14932
//  後処理 Clock :                      2786

#include "Pre.h"

// include ↓↓↓↓↓↓↓↓↓↓↓↓↓↓↓
#include <math.h>
// include ↑↑↑↑↑↑↑↑↑↑↑↑↑↑↑

// Config ↓↓↓↓↓↓↓↓↓↓↓↓↓↓↓

// TITANIC か HOUSE_PRICES を選択
const int skipNum = 5;  // TITANIC
const unsigned int threadNum = 891;  // train data num // TITANIC

//const int skipNum = 3; // HOUSE PRICES
//const unsigned int threadNum = 1392; // train data num // HOUSE PRICES

const int neuralNetNum = 420;
const int inputNeuronNumMax = 20;

const int middleLayerCountMax = 10;
const int middleLayerNeuronNumMax = 40;

// FloatShareNum
#define FS1     11
#define FS2     1 // dummy
#define FS3     1 // dummy
#define FS4     (inputNeuronNumMax)
#define FS5     1 // trainDataNumMax
#define FS6     1 // dummy
#define FS7     middleLayerCountMax

#define FS8     inputNeuronNumMax
#define FS9     ((1 * inputNeuronNumMax) * skipNum)
#define FS10    1 // dummy
#define FS11    (inputNeuronNumMax * skipNum)

#define FS12    (middleLayerNeuronNumMax * middleLayerCountMax)
#define FS13    (((middleLayerNeuronNumMax * middleLayerNeuronNumMax) * middleLayerCountMax) * skipNum)
#define FS14    1 // dummy
#define FS15    ((middleLayerNeuronNumMax * middleLayerCountMax) * skipNum)

#define FS16    1
#define FS17    ((middleLayerNeuronNumMax * 1) * skipNum)
#define FS18    1 // dummy
#define FS19    (1 * skipNum)

// FloatInput0Num
#define FIZ1    skipNum  // nnNo

#define FIZ2    ((1 * inputNeuronNumMax) * skipNum)  // input layer W
#define FIZ3    (((middleLayerNeuronNumMax * middleLayerNeuronNumMax) * middleLayerCountMax) * skipNum)  // middle layer W
#define FIZ4    ((middleLayerNeuronNumMax * 1) * skipNum)  // output layer W

// FloatInputNum
//#define FI1    1

// FloatOutputNum
#define FO1    neuralNetNum // PredictValueList

// Offset(FloatShare)
#define FSOS1     FS1
#define FSOS2     (FSOS1 + FS2)
#define FSOS3     (FSOS2 + FS3)
#define FSOS4     (FSOS3 + FS4)
#define FSOS5     (FSOS4 + FS5)
#define FSOS6     (FSOS5 + FS6)
#define FSOS7     (FSOS6 + FS7)
#define FSOS8     (FSOS7 + FS8)
#define FSOS9     (FSOS8 + FS9)
#define FSOS10    (FSOS9 + FS10)
#define FSOS11    (FSOS10 + FS11)
#define FSOS12    (FSOS11 + FS12)
#define FSOS13    (FSOS12 + FS13)
#define FSOS14    (FSOS13 + FS14)
#define FSOS15    (FSOS14 + FS15)
#define FSOS16    (FSOS15 + FS16)
#define FSOS17    (FSOS16 + FS17)
#define FSOS18    (FSOS17 + FS18)
#define FSOS19    (FSOS18 + FS19)

// Offset(FloatInput0)
#define FIZOS1     FIZ1
#define FIZOS2     (FIZOS1 + FIZ2)
#define FIZOS3     (FIZOS2 + FIZ3)
#define FIZOS4     (FIZOS3 + FIZ4)

// Offset(FloatInput)
//#define FIOS1     FI1

// Offset(FloatOutput)
#define FOOS1     FO1

const unsigned int fShareNum = FSOS19;
const unsigned int fInput0Num = FIZOS4;
const unsigned int fInputNum = 0;
//const unsigned int fWorkNum = 1;
const unsigned int fOutputNum = FOOS1;

#define fsCorrectNum             FS(0)
#define fsAccuracyRate           FS(1)
#define fsCalcError              FS(2)
#define fsTrainDataNum           FS(3)
#define fsAiType                 FS(4)
#define fsThresholdVal           FS(5)
#define fsPosiValue              FS(6)
#define fsInputNeuronNum         FS(7)
#define fsMiddleLayerCount       FS(8)
#define fsOutputNeuronNum        FS(9)    // 1
#define fsOutputValue            FS(10)

#define fsInputValueList(aryIdx)             FS(FSOS3 + (aryIdx))
#define fsCorrectValueList(aryIdx)           FS(FSOS4 + (aryIdx))
#define fsMiddleNeuronNums(aryIdx)           FS(FSOS6 + (aryIdx))

#define fsInputLayerDendriteNum(aryIdx)      FS(FSOS7 + (aryIdx))
#define fsInputLayerXVal(skipIdx, aryIdx)    FS(FSOS8 + ((1 * inputNeuronNumMax) * skipIdx + aryIdx))
#define fsInputLayerYVal(skipIdx, aryIdx)    FS(FSOS10 + (inputNeuronNumMax * skipIdx + aryIdx))

#define fsMiddleLayerDendriteNum(layIdx, aryIdx)            FS(FSOS11 + (layIdx) * middleLayerNeuronNumMax + (aryIdx))
#define fsMiddleLayerXVal(skipIdx, layIdx, aryIdx, dIdx)    FS(FSOS12 + (((middleLayerNeuronNumMax * middleLayerNeuronNumMax) * middleLayerCountMax) * skipIdx) + (layIdx) * middleLayerNeuronNumMax * middleLayerNeuronNumMax + (aryIdx) * middleLayerNeuronNumMax + (dIdx))
#define fsMiddleLayerYVal(skipIdx, layIdx, aryIdx)          FS(FSOS14 + ((middleLayerNeuronNumMax * middleLayerCountMax) * skipIdx) + (layIdx) * middleLayerNeuronNumMax + (aryIdx))

#define fsOutputLayerDendriteNum(aryIdx)            FS(FSOS15 + (aryIdx))
#define fsOutputLayerXVal(skipIdx, aryIdx, dIdx)    FS(FSOS16 + ((middleLayerNeuronNumMax * 1) * skipIdx) + (aryIdx) * middleLayerNeuronNumMax + (dIdx))
#define fsOutputLayerYVal(skipIdx, aryIdx)          FS(FSOS18 + (1 * skipIdx) + (aryIdx)) // 1個

#define fizNnNo(skipIdx)                      FI0(skipIdx)

#define fizInputLayerWVal(skipIdx, aryIdx)                  FI0(FIZOS1 + ((1 * inputNeuronNumMax) * skipIdx + aryIdx))
#define fizMiddleLayerWVal(skipIdx, layIdx, aryIdx, dIdx)   FI0(FIZOS2 + (((middleLayerNeuronNumMax * middleLayerNeuronNumMax) * middleLayerCountMax) * skipIdx) + (layIdx) * middleLayerNeuronNumMax * middleLayerNeuronNumMax + (aryIdx) * middleLayerNeuronNumMax + (dIdx))
#define fizOutputLayerWVal(skipIdx, aryIdx, dIdx)           FI0(FIZOS3 + ((middleLayerNeuronNumMax * 1) * skipIdx) + (aryIdx) * middleLayerNeuronNumMax + (dIdx))

#define fiOne           FI(0)

#define fwOne           FW(0)

#define foPredictValueList(aryIdx)           FO(0 + (aryIdx))

// Config ↑↑↑↑↑↑↑↑↑↑↑↑↑↑↑

#include "Export.h"

// 並列処理 ↓↓↓↓↓↓↓↓↓↓↓↓↓↓

// FuncKernel関数から呼ばれる関数 ↓↓↓↓↓↓↓↓↓↓↓↓↓↓

//__device__ void ComputeOutputValue()
//{
//
//}

// FuncKernel関数から呼ばれる関数 ↑↑↑↑↑↑↑↑↑↑↑↑↑↑

__global__ void FuncKernel(float* devFloatOutputArray, float* devFloatInput0Array, float* devFloatInputArray, float* devFloatWorkArray, float* devFloatShareArray)
{
    int bIdx = blockIdx.x;
    int tIdx = threadIdx.x;  // threadは1024まで
    //long long startClock;
//    __shared__ float inputArrayShare[inputNum];  // MAX 0xC000 byte(49KB)

    //ComputeOutputValue();

    //__syncthreads();  // 同期化

    // ここに並列処理のコードを書く ↓↓↓↓↓↓↓↓↓↓↓↓↓↓
    {
        int inputNeuronNum = fsInputNeuronNum;
        int nnNo = (int)(fizNnNo(tIdx));

        // 開始時間を記録
        //startClock = clock64();

        // 入力層にパラメータを渡す
        for (int i = 0; i < inputNeuronNum; i++)
        {
            fsInputLayerXVal(tIdx, i) = fsInputValueList(i);
        }

        //printf("入力層にパラメータを渡す Clock : %lld\n", clock64() - startClock);  // 19753

        //startClock = clock64();

        // 出力値を計算する
        {
            //startClock = clock64();

            // 入力層の各ニューロンの y の値を計算する。
            for (int row = 0; row < inputNeuronNum; row++)
            {
                float yVal = 0;

                // ｙ ＝ ｘ0ｗ0 ＋ ｘ1ｗ1 ＋ ｘ2ｗ2 ＋ ｘ3ｗ3
                yVal += fsInputLayerXVal(tIdx, row) * fizInputLayerWVal(tIdx, row);

                fsInputLayerYVal(tIdx, row) = yVal;

                //printf("fsInputLayerYVal(tIdx, row) = %f\n", fsInputLayerYVal(tIdx, row));
            }

            //printf("入力層の各ニューロンの y の値を計算する。 Clock : %lld\n", clock64() - startClock);  // 19815

            //startClock = clock64();  // ここが一番時間がかかる

            //printf("fsMiddleLayerCount = %f\n", fsMiddleLayerCount);

            // 中間層の各ニューロンに前列から y の値を伝達させつつ、
            // 各ニューロンの y の値を計算する。
            {
                int middleLayerCount = fsMiddleLayerCount;

                for (int c = 0; c < middleLayerCount; c++)
                {
                    int middleNeuronNumsC = fsMiddleNeuronNums(c);

                    if (c == 0)
                    {
                        for (int row = 0; row < middleNeuronNumsC; row++)
                        {
                            // このようなintへの置き換えは処理速度向上効果あり
                            int middleLayerDendriteNumCRow = fsMiddleLayerDendriteNum(c, row);

                            for (int x = 0; x < middleLayerDendriteNumCRow; x++)
                            {
                                fsMiddleLayerXVal(tIdx, c, row, x) = fsInputLayerYVal(tIdx, x);
                            }

                            //middleLayer[c].neurons[r].ComputeYVal();
                            {
                                float yVal = 0;

                                // ｙ ＝ ｘ0ｗ0 ＋ ｘ1ｗ1 ＋ ｘ2ｗ2 ＋ ｘ3ｗ3
                                for (int i = 0; i < middleLayerDendriteNumCRow; i++)
                                {
                                    yVal += fsMiddleLayerXVal(tIdx, c, row, i) * fizMiddleLayerWVal(tIdx, c, row, i);
                                }

                                fsMiddleLayerYVal(tIdx, c, row) = yVal;

                                //printf("c = %d   row = %d   fsMiddleLayerYVal(tIdx, c, row) = %f\n", c, row, fsMiddleLayerYVal(tIdx, c, row));
                            }
                        }
                    }
                    else
                    {
                        //printf("middleNeuronNumsC = %f\n", middleNeuronNumsC);

                        for (int row = 0; row < middleNeuronNumsC; row++)
                        {
                            int middleLayerDendriteNumCRow = fsMiddleLayerDendriteNum(c, row);

                            //printf("middleLayerDendriteNumCRow = %d\n", middleLayerDendriteNumCRow);

                            for (int x = 0; x < middleLayerDendriteNumCRow; x++)
                            {
                                //printf("c - 1 = %d   x = %d   fsMiddleLayerYVal(tIdx, c - 1, x) = %f\n", c - 1, x, fsMiddleLayerYVal(tIdx, c - 1, x));

                                fsMiddleLayerXVal(tIdx, c, row, x) = fsMiddleLayerYVal(tIdx, c - 1, x);

                                //printf("c = %d   row = %d   x = %d   fsMiddleLayerXVal(tIdx, c, row, x) = %f\n", c, row, x, fsMiddleLayerXVal(tIdx, c, row, x));
                            }

                            //middleLayer[c].neurons[r].ComputeYVal();
                            {
                                float yVal = 0;

                                // ｙ ＝ ｘ0ｗ0 ＋ ｘ1ｗ1 ＋ ｘ2ｗ2 ＋ ｘ3ｗ3
                                for (int i = 0; i < middleLayerDendriteNumCRow; i++)
                                {
                                    //printf("c = %d   row = %d   i = %d   fsMiddleLayerXVal(tIdx, c, row, i) = %f\n", c, row, i, fsMiddleLayerXVal(tIdx, c, row, i));
                                    //printf("fizMiddleLayerWVal(tIdx, c, row, i) = %f\n", fizMiddleLayerWVal(tIdx, c, row, i));

                                    yVal += fsMiddleLayerXVal(tIdx, c, row, i) * fizMiddleLayerWVal(tIdx, c, row, i);
                                }

                                fsMiddleLayerYVal(tIdx, c, row) = yVal;

                                //printf("c = %d   row = %d   fsMiddleLayerYVal(tIdx, c, row) = %f\n", c, row, fsMiddleLayerYVal(tIdx, c, row));
                            }
                        }
                    }
                }
            }

            // ここが一番時間がかかる
            //printf("中間層の各ニューロンに前列から y の値を伝達させつつ Clock : %lld\n", clock64() - startClock);  // 1995285 => 8789465

            //startClock = clock64();

            {
                int outputLayerDendriteNum0 = fsOutputLayerDendriteNum(0);

                // 中間層の最後の列から y の値を出力層のニューロンに伝達させる。
                for (int x = 0; x < outputLayerDendriteNum0; x++)
                {
                    fsOutputLayerXVal(tIdx, 0, x) = fsMiddleLayerYVal(tIdx, (int)fsMiddleLayerCount - 1, x);

                    //printf("x = %d   fsMiddleLayerYVal(tIdx, (int)fsMiddleLayerCount - 1, x) = %f\n", x, fsMiddleLayerYVal(tIdx, (int)fsMiddleLayerCount - 1, x));
                    //printf("fsOutputLayerXVal(tIdx, 0, x) = %f\n", fsOutputLayerXVal(tIdx, 0, x));
                }

                //printf("y の値を出力層のニューロンに伝達させる Clock : %lld\n", clock64() - startClock);  // 16522

                //startClock = clock64();

                // 出力層のニューロンの y の値を計算する。
                //outputLayer.ComputeYVal();
                {
                    float yVal = 0;

                    // ｙ ＝ ｘ0ｗ0 ＋ ｘ1ｗ1 ＋ ｘ2ｗ2 ＋ ｘ3ｗ3
                    for (int i = 0; i < outputLayerDendriteNum0; i++)
                    {
                        yVal += fsOutputLayerXVal(tIdx, 0, i) * fizOutputLayerWVal(tIdx, 0, i);
                    }

                    fsOutputLayerYVal(tIdx, 0) = yVal;
                }
            }

            //printf("出力層のニューロンの y の値を計算する Clock : %lld\n", clock64() - startClock);  // 14932
        }

        //printf("出力値を計算する Clock : %lld\n", clock64() - startClock);  // 2042214

        //startClock = clock64();

        // 出力値を取得する
        foPredictValueList(nnNo) = fsOutputLayerYVal(tIdx, 0);
        //printf("foPredictValueList(nnNo) = %f\n", foPredictValueList(nnNo));

        //printf("後処理 Clock : %lld\n", clock64() - startClock);  // 2786

        // 終了時間を記録
        //printf("終了時間 Clock : %lld\n", clock64() - startClock);  // 2907097002
    }
    // ここに並列処理のコードを書く ↑↑↑↑↑↑↑↑↑↑↑↑↑↑
}

// 並列処理 ↑↑↑↑↑↑↑↑↑↑↑↑↑↑

#include "Post.h"
